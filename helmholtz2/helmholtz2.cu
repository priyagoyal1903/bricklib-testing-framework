#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void helmholtz2_naive(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];

    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
// $END naive

// $START codegen
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]

__global__ void helmholtz2_codegen(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
// $END codegen

// $START naive-bricks
__global__ void helmholtz2_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;

    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];

    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
// $END naive-bricks

// $START codegen-bricks
__global__ void helmholtz2_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks
