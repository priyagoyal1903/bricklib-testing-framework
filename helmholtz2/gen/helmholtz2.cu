#include "hip/hip_runtime.h"
# 1 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./helmholtz2.h"
#include <brick-hip.h>

__global__ void helmholtz2_naive2(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen2(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
# 1 "VSTile-helmholtz22.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = c[0] * alpha(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) - c[1] * c[2] * (beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + -1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + -1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + -1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2));
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 35 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-helmholtz22.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_beta_i000_vecbuf;
      bElem _cg_beta_i100_vecbuf;
      bElem _cg_x000_vecbuf;
      bElem _cg_x100_vecbuf;
      bElem _cg_x_100_vecbuf;
      bElem _cg_beta_j000_vecbuf;
      bElem _cg_beta_j010_vecbuf;
      bElem _cg_beta_k000_vecbuf;
      bElem _cg_beta_k001_vecbuf;
      {
        // New offset [-1, 0, -1]
        bElem _cg_beta_k101_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor4 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[1] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, -1, 1]
            bElem _cg_beta_j010_reg;
            bElem _cg_x000_reg;
            {
              _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor10 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x000_reg;
            bElem _cg_x100_reg;
            bElem _cg_beta_j100_reg;
            bElem _cg_beta_k101_reg;
            bElem _cg_beta_k100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
              _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 128 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
              bElem _cg_vectmp0;
              // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_x000_reg = _cg_vectmp0;
              _cg_x100_reg = _cg_x000_vecbuf;
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_j100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k101_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k100_reg * _cg_x100_reg;
            buf0[0 + rel] += _cg_beta_k100_reg * _cg_x100_reg;
            buf0[2 + rel] += _cg_beta_k101_reg * _cg_x100_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x100_reg;
            bElem _cg_x000_reg;
            bElem _cg_beta_j010_reg;
            {
              _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
              dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_beta_i100_reg = _cg_vectmp2;
              bElem _cg_vectmp3;
              // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
              dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_x100_reg = _cg_vectmp3;
              _cg_x000_reg = _cg_x000_vecbuf;
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_beta_j100_reg;
            bElem _cg_x100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor16 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_x100_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j100_reg * _cg_x100_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 448 + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor22 * beta_k.step + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[6] += _cg_beta_k100_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_beta_k100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_x000_vecbuf = x.dat[neighbor22 * x.step + hipThreadIdx_x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_k100_reg * _cg_x100_reg;
      }
    }
  }
  bElem buf1[8];
  {
    {
      bElem _cg_alpha000_vecbuf;
      bElem _cg_x000_vecbuf;
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            bElem _cg_alpha000_reg;
            bElem _cg_x000_reg;
            {
              _cg_alpha000_vecbuf = alpha.dat[neighbor13 * alpha.step + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + (hipThreadIdx_x + rel * 64)];
              _cg_alpha000_reg = _cg_alpha000_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf1[0 + rel] = c[0] * _cg_alpha000_reg * _cg_x000_reg - c[1] * c[2] * buf0[0 + rel];
          }
          _cg_rel2 += 1;
        }
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf1[sti];
    }
  }
}
# 63 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
__global__ void helmholtz2_naive3(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen3(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
# 1 "VSTile-helmholtz23.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = c[0] * alpha(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) - c[1] * c[2] * (beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + -1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + -1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + -1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2));
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 92 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-helmholtz23.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_beta_i000_vecbuf;
      bElem _cg_beta_i100_vecbuf;
      bElem _cg_x000_vecbuf;
      bElem _cg_x100_vecbuf;
      bElem _cg_x_100_vecbuf;
      bElem _cg_beta_j000_vecbuf;
      bElem _cg_beta_j010_vecbuf;
      bElem _cg_beta_k000_vecbuf;
      bElem _cg_beta_k001_vecbuf;
      {
        // New offset [-1, 0, -1]
        bElem _cg_beta_k101_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor4 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[1] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, -1, 1]
            bElem _cg_beta_j010_reg;
            bElem _cg_x000_reg;
            {
              _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor10 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x000_reg;
            bElem _cg_x100_reg;
            bElem _cg_beta_j100_reg;
            bElem _cg_beta_k101_reg;
            bElem _cg_beta_k100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
              _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 128 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
              bElem _cg_vectmp0;
              // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_x000_reg = _cg_vectmp0;
              _cg_x100_reg = _cg_x000_vecbuf;
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_j100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k101_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k100_reg * _cg_x100_reg;
            buf0[0 + rel] += _cg_beta_k100_reg * _cg_x100_reg;
            buf0[2 + rel] += _cg_beta_k101_reg * _cg_x100_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x100_reg;
            bElem _cg_x000_reg;
            bElem _cg_beta_j010_reg;
            {
              _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
              dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_beta_i100_reg = _cg_vectmp2;
              bElem _cg_vectmp3;
              // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
              dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_x100_reg = _cg_vectmp3;
              _cg_x000_reg = _cg_x000_vecbuf;
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_beta_j100_reg;
            bElem _cg_x100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor16 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_x100_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j100_reg * _cg_x100_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 448 + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor22 * beta_k.step + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[6] += _cg_beta_k100_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_beta_k100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_x000_vecbuf = x.dat[neighbor22 * x.step + hipThreadIdx_x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_k100_reg * _cg_x100_reg;
      }
    }
  }
  bElem buf1[8];
  {
    {
      bElem _cg_alpha000_vecbuf;
      bElem _cg_x000_vecbuf;
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            bElem _cg_alpha000_reg;
            bElem _cg_x000_reg;
            {
              _cg_alpha000_vecbuf = alpha.dat[neighbor13 * alpha.step + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + (hipThreadIdx_x + rel * 64)];
              _cg_alpha000_reg = _cg_alpha000_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf1[0 + rel] = c[0] * _cg_alpha000_reg * _cg_x000_reg - c[1] * c[2] * buf0[0 + rel];
          }
          _cg_rel2 += 1;
        }
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf1[sti];
    }
  }
}
# 120 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
__global__ void helmholtz2_naive5(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen5(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
# 1 "VSTile-helmholtz25.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = c[0] * alpha(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) - c[1] * c[2] * (beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x + 1, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x + -1, j + _cg_idx1, k + _cg_idx2) - beta_i(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1 + -1, k + _cg_idx2) - beta_j(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + 1) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) + beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2 + -1) - beta_k(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2) * x(i + hipThreadIdx_x, j + _cg_idx1, k + _cg_idx2));
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 149 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-helmholtz25.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_beta_i000_vecbuf;
      bElem _cg_beta_i100_vecbuf;
      bElem _cg_x000_vecbuf;
      bElem _cg_x100_vecbuf;
      bElem _cg_x_100_vecbuf;
      bElem _cg_beta_j000_vecbuf;
      bElem _cg_beta_j010_vecbuf;
      bElem _cg_beta_k000_vecbuf;
      bElem _cg_beta_k001_vecbuf;
      {
        // New offset [-1, 0, -1]
        bElem _cg_beta_k101_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor4 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 64 + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[1] += _cg_beta_k101_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[0] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[0] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[0] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[0] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, -1, 1]
            bElem _cg_beta_j010_reg;
            bElem _cg_x000_reg;
            {
              _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor10 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x000_reg;
            bElem _cg_x100_reg;
            bElem _cg_beta_j100_reg;
            bElem _cg_beta_k101_reg;
            bElem _cg_beta_k100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
              _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_k001_vecbuf = beta_k.dat[neighbor13 * beta_k.step + 128 + (hipThreadIdx_x + rel * 64)];
              _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
              bElem _cg_vectmp0;
              // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_x000_reg = _cg_vectmp0;
              _cg_x100_reg = _cg_x000_vecbuf;
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
              _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_j100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k101_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_k100_reg * _cg_x100_reg;
            buf0[0 + rel] += _cg_beta_k100_reg * _cg_x100_reg;
            buf0[2 + rel] += _cg_beta_k101_reg * _cg_x100_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_beta_i100_reg;
            bElem _cg_x100_reg;
            bElem _cg_x000_reg;
            bElem _cg_beta_j010_reg;
            {
              _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
              dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_beta_i100_reg = _cg_vectmp2;
              bElem _cg_vectmp3;
              // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
              dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_x100_reg = _cg_vectmp3;
              _cg_x000_reg = _cg_x000_vecbuf;
              _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_i100_reg * _cg_x100_reg;
            buf0[1 + rel] -= _cg_beta_i100_reg * _cg_x000_reg;
            buf0[1 + rel] -= _cg_beta_j010_reg * _cg_x000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_beta_j100_reg;
            bElem _cg_x100_reg;
            {
              _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = x.dat[neighbor16 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = x.dat[neighbor13 * x.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
              dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
              _cg_x100_reg = _cg_x000_vecbuf;
            }
            buf0[1 + rel] += _cg_beta_j100_reg * _cg_x100_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_beta_j010_reg;
        bElem _cg_x000_reg;
        {
          _cg_beta_j010_vecbuf = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor10 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
          _cg_x000_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x000_reg;
        bElem _cg_x100_reg;
        bElem _cg_beta_j100_reg;
        bElem _cg_beta_k101_reg;
        bElem _cg_beta_k100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_beta_i000_vecbuf = beta_i.dat[neighbor13 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x_100_vecbuf = x.dat[neighbor12 * x.step + 448 + hipThreadIdx_x];
          _cg_x000_vecbuf = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          _cg_beta_k001_vecbuf = beta_k.dat[neighbor22 * beta_k.step + hipThreadIdx_x];
          _cg_beta_i100_reg = _cg_beta_i000_vecbuf;
          bElem _cg_vectmp0;
          // merge0 _cg_x_100_vecbuf ,_cg_x000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_x_100_vecbuf, _cg_x000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_x000_reg = _cg_vectmp0;
          _cg_x100_reg = _cg_x000_vecbuf;
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_beta_k101_reg = _cg_beta_k001_vecbuf;
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_j100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k101_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_k100_reg * _cg_x100_reg;
        buf0[6] += _cg_beta_k100_reg * _cg_x100_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_beta_i100_reg;
        bElem _cg_x100_reg;
        bElem _cg_x000_reg;
        bElem _cg_beta_j010_reg;
        {
          _cg_beta_i100_vecbuf = beta_i.dat[neighbor14 * beta_i.step + 448 + hipThreadIdx_x];
          _cg_x100_vecbuf = x.dat[neighbor14 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          _cg_vectmp0 = beta_j.dat[neighbor16 * beta_j.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = beta_j.dat[neighbor13 * beta_j.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_beta_j010_vecbuf
          dev_shl(_cg_beta_j010_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_beta_i000_vecbuf ,_cg_beta_i100_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_beta_i000_vecbuf, _cg_beta_i100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_beta_i100_reg = _cg_vectmp2;
          bElem _cg_vectmp3;
          // merge0 _cg_x000_vecbuf ,_cg_x100_vecbuf, 1 -> _cg_vectmp3
          dev_shl(_cg_vectmp3, _cg_x000_vecbuf, _cg_x100_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_x100_reg = _cg_vectmp3;
          _cg_x000_reg = _cg_x000_vecbuf;
          _cg_beta_j010_reg = _cg_beta_j010_vecbuf;
        }
        buf0[7] += _cg_beta_i100_reg * _cg_x100_reg;
        buf0[7] -= _cg_beta_i100_reg * _cg_x000_reg;
        buf0[7] -= _cg_beta_j010_reg * _cg_x000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_beta_j100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_j000_vecbuf = _cg_beta_j010_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = x.dat[neighbor16 * x.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = x.dat[neighbor13 * x.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_x000_vecbuf
          dev_shl(_cg_x000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_beta_j100_reg = _cg_beta_j000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_j100_reg * _cg_x100_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_beta_k100_reg;
        bElem _cg_x100_reg;
        {
          _cg_beta_k000_vecbuf = _cg_beta_k001_vecbuf;
          _cg_x000_vecbuf = x.dat[neighbor22 * x.step + hipThreadIdx_x];
          _cg_beta_k100_reg = _cg_beta_k000_vecbuf;
          _cg_x100_reg = _cg_x000_vecbuf;
        }
        buf0[7] += _cg_beta_k100_reg * _cg_x100_reg;
      }
    }
  }
  bElem buf1[8];
  {
    {
      bElem _cg_alpha000_vecbuf;
      bElem _cg_x000_vecbuf;
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            bElem _cg_alpha000_reg;
            bElem _cg_x000_reg;
            {
              _cg_alpha000_vecbuf = alpha.dat[neighbor13 * alpha.step + (hipThreadIdx_x + rel * 64)];
              _cg_x000_vecbuf = x.dat[neighbor13 * x.step + (hipThreadIdx_x + rel * 64)];
              _cg_alpha000_reg = _cg_alpha000_vecbuf;
              _cg_x000_reg = _cg_x000_vecbuf;
            }
            buf1[0 + rel] = c[0] * _cg_alpha000_reg * _cg_x000_reg - c[1] * c[2] * buf0[0 + rel];
          }
          _cg_rel2 += 1;
        }
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf1[sti];
    }
  }
}
# 177 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz2.cu" 2

}
