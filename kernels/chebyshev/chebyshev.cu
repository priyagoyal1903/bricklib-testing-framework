#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void chebyshev_naive(bElem (*Ac)[STRIDE1][STRIDE0], bElem (*Ap)[STRIDE1][STRIDE0], bElem (*Dinv)[STRIDE1][STRIDE0], bElem (*RHS)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
  unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
  unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
  unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

  out[k][j][i] = 
    Ac[k][j][i] + c[0] * (Ac[k][j][i] + Ap[k][j][i]) +
    c[1] * Dinv[k][j][i] *
    (RHS[k][j][i] +
    (Ac[k][j][i] + 
    c[2] * 
      (
        0.03 * (Ac[k - 1][j - 1][i - 1] + Ac[k - 1][j - 1][i + 1] +
                Ac[k - 1][j + 1][i - 1] + Ac[k - 1][j + 1][i + 1] +
                Ac[k + 1][j - 1][i - 1] + Ac[k + 1][j - 1][i + 1] +
                Ac[k + 1][j + 1][i - 1] + Ac[k + 1][j + 1][i + 1]) +
        0.1 * (Ac[k - 1][j - 1][i] + Ac[k - 1][j][i - 1] +
               Ac[k - 1][j][i + 1] + Ac[k - 1][j + 1][i] +
               Ac[k][j - 1][i - 1] + Ac[k][j - 1][i + 1] +
               Ac[k][j + 1][i - 1] + Ac[k][j + 1][i + 1] +
               Ac[k + 1][j - 1][i] + Ac[k + 1][j][i - 1] +
               Ac[k + 1][j][i + 1] + Ac[k + 1][j + 1][i]) +
        0.46 * (Ac[k - 1][j][i] + Ac[k][j - 1][i] + Ac[k][j][i - 1] +
                Ac[k + 1][j][i] + Ac[k][j + 1][i] + Ac[k][j][i + 1]) +
        4.26 * Ac[k][j][i]
               )));
}
// $END naive

// $START naive-bricks
__global__ void chebyshev_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  unsigned i = threadIdx.x;
  unsigned j = threadIdx.y;
  unsigned k = threadIdx.z;

  out[b][k][j][i] = 
    Ac[b][k][j][i] + c[0] * (Ac[b][k][j][i] + Ap[b][k][j][i]) +
    c[1] * Dinv[b][k][j][i] *
    (RHS[b][k][j][i] +
    (Ac[b][k][j][i] + 
    c[2] * 
      (
        0.03 * (Ac[b][k - 1][j - 1][i - 1] + Ac[b][k - 1][j - 1][i + 1] +
                Ac[b][k - 1][j + 1][i - 1] + Ac[b][k - 1][j + 1][i + 1] +
                Ac[b][k + 1][j - 1][i - 1] + Ac[b][k + 1][j - 1][i + 1] +
                Ac[b][k + 1][j + 1][i - 1] + Ac[b][k + 1][j + 1][i + 1]) +
        0.1 * (Ac[b][k - 1][j - 1][i] + Ac[b][k - 1][j][i - 1] +
               Ac[b][k - 1][j][i + 1] + Ac[b][k - 1][j + 1][i] +
               Ac[b][k][j - 1][i - 1] + Ac[b][k][j - 1][i + 1] +
               Ac[b][k][j + 1][i - 1] + Ac[b][k][j + 1][i + 1] +
               Ac[b][k + 1][j - 1][i] + Ac[b][k + 1][j][i - 1] +
               Ac[b][k + 1][j][i + 1] + Ac[b][k + 1][j + 1][i]) +
        0.46 * (Ac[b][k - 1][j][i] + Ac[b][k][j - 1][i] + Ac[b][k][j][i - 1] +
                Ac[b][k + 1][j][i] + Ac[b][k][j + 1][i] + Ac[b][k][j][i + 1]) +
        4.26 * Ac[b][k][j][i]
               )));
}
}
// $END naive-bricks

// $START codegen-bricks
__global__ void chebyshev_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks

// $START codegen
#define Ac(a, b, c) Ac_arr[c][b][a]
#define Ap(a, b, c) Ap_arr[c][b][a]
#define Dinv(a, b, c) Dinv_arr[c][b][a]
#define RHS(a, b, c) RHS_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]

__global__ void chebyshev_codegen(bElem (*Ac_arr)[STRIDE1][STRIDE0], bElem (*Ap_arr)[STRIDE1][STRIDE0], bElem (*Dinv_arr)[STRIDE1][STRIDE0], bElem (*RHS_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
  long k = OFF2 + (blockIdx.z * TILE2);
  long j = OFF1 + (blockIdx.y * TILE1);
  long i = OFF0 + (blockIdx.x * TILE0);
  tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef Ac
#undef Ap
#undef Dinv
#undef RHS
#undef out
// $END codegen

