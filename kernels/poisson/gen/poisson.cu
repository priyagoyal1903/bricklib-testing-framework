#include "hip/hip_runtime.h"
# 1 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./poisson.h"
#include <brick-hip.h>

__global__ void poisson_naive2(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen2(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
# 1 "VSTile-poisson2.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = 0;
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      {
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + -1);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x, j, k + -1);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + -1);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + -1);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + -1);
      }
      {
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k);
      }
      {
        buf0[0] += 2.666 * in(i + hipThreadIdx_x, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[1] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[9] -= 0.0833 * in(i + hipThreadIdx_x, j, k);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] += 2.666 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[6] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[14] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k);
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k);
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
          }
          {
            buf0[8 + rel] += 2.666 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
          }
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[9 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[18 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
              }
              _cg_rel1 += 1;
            }
          }
          {
            buf0[15 + rel] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[22 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[6 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
          }
          {
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
          }
          _cg_rel2 += 8;
        }
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 7);
      }
      {
        buf0[56] += 2.666 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[57] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[49] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 7);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[50 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[48 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[62] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[54] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 7);
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 7);
      }
      {
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 8);
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 8);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 8);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 8);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 8);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 8);
      }
      {
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 8);
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 29 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
#undef in
#undef out
__global__ void poisson_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-poisson2.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_in000_vecbuf;
      bElem _cg_in_100_vecbuf;
      {
        // New offset [0, -1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor1 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor3 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor5 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor7 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, -1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] += 2.666 * _cg_in000_reg;
        buf0[1] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [-1, -1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor9 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor10 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor11 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] += 2.666 * _cg_in000_reg;
            buf0[2 + rel] -= 0.166 * _cg_in000_reg;
            buf0[0 + rel] -= 0.166 * _cg_in000_reg;
          }
          {
            // New offset [1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor15 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor16 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor17 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [-1, -1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] += 2.666 * _cg_in000_reg;
        buf0[6] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor19 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor21 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor23 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor25 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf0[sti];
    }
  }
}
# 51 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
__global__ void poisson_naive3(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen3(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
# 1 "VSTile-poisson3.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = 0;
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      {
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + -1);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x, j, k + -1);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + -1);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + -1);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + -1);
      }
      {
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k);
      }
      {
        buf0[0] += 2.666 * in(i + hipThreadIdx_x, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[1] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[9] -= 0.0833 * in(i + hipThreadIdx_x, j, k);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] += 2.666 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[6] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[14] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k);
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k);
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
          }
          {
            buf0[8 + rel] += 2.666 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
          }
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[9 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[18 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
              }
              _cg_rel1 += 1;
            }
          }
          {
            buf0[15 + rel] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[22 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[6 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
          }
          {
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
          }
          _cg_rel2 += 8;
        }
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 7);
      }
      {
        buf0[56] += 2.666 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[57] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[49] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 7);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[50 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[48 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[62] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[54] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 7);
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 7);
      }
      {
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 8);
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 8);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 8);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 8);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 8);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 8);
      }
      {
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 8);
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 74 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
#undef in
#undef out
__global__ void poisson_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-poisson3.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_in000_vecbuf;
      bElem _cg_in_100_vecbuf;
      {
        // New offset [0, -1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor1 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor3 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor5 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor7 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, -1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] += 2.666 * _cg_in000_reg;
        buf0[1] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [-1, -1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor9 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor10 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor11 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] += 2.666 * _cg_in000_reg;
            buf0[2 + rel] -= 0.166 * _cg_in000_reg;
            buf0[0 + rel] -= 0.166 * _cg_in000_reg;
          }
          {
            // New offset [1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor15 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor16 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor17 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [-1, -1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] += 2.666 * _cg_in000_reg;
        buf0[6] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor19 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor21 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor23 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor25 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf0[sti];
    }
  }
}
# 96 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
__global__ void poisson_naive5(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen5(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
# 1 "VSTile-poisson5.py-HIP-8x8x64" 1
{
  bElem buf0[64];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[0 + rel] = 0;
              }
              _cg_rel1 += 1;
            }
          }
          _cg_rel2 += 8;
        }
      }
    }
    {
      {
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + -1);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + -1);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x, j, k + -1);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + -1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + -1);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + -1);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + -1);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + -1);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + -1);
      }
      {
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + -1);
      }
      {
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k);
        buf0[0] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k);
      }
      {
        buf0[0] += 2.666 * in(i + hipThreadIdx_x, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[0] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[8] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[1] -= 0.166 * in(i + hipThreadIdx_x, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k);
        buf0[1] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k);
        buf0[9] -= 0.0833 * in(i + hipThreadIdx_x, j, k);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[1 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k);
            buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[7] += 2.666 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[6] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k);
        buf0[6] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k);
        buf0[14] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k);
      }
      {
        buf0[7] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k);
        buf0[7] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k);
        buf0[15] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k);
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + _cg_idx2 + 1);
          }
          {
            buf0[8 + rel] += 2.666 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + _cg_idx2 + 1);
            buf0[9 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + _cg_idx2 + 1);
            buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
            buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j, k + _cg_idx2 + 1);
          }
          {
            long _cg_rel1 = rel;
            for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
            {
              long rel = _cg_rel1;
              {
                buf0[9 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[9 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[17 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[1 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[10 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[8 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[18 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[2 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[16 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
                buf0[0 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + _cg_idx2 + 1);
              }
              _cg_rel1 += 1;
            }
          }
          {
            buf0[15 + rel] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + _cg_idx2 + 1);
            buf0[14 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + _cg_idx2 + 1);
            buf0[22 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
            buf0[6 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + _cg_idx2 + 1);
          }
          {
            buf0[15 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + _cg_idx2 + 1);
            buf0[15 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + _cg_idx2 + 1);
            buf0[23 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
            buf0[7 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + _cg_idx2 + 1);
          }
          _cg_rel2 += 8;
        }
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + -1, k + 7);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + -1, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 7);
      }
      {
        buf0[56] += 2.666 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[48] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[57] -= 0.166 * in(i + hipThreadIdx_x, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 7);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 7);
        buf0[49] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 7);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] += 2.666 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[49 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 7);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 7);
            buf0[50 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
            buf0[48 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 7);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] += 2.666 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[62] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 7);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 7);
        buf0[54] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 7);
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 8, k + 7);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 8, k + 7);
        buf0[55] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 7);
      }
      {
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x, j + -1, k + 8);
      }
      {
        buf0[56] -= 0.166 * in(i + hipThreadIdx_x, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + -1, j, k + 8);
        buf0[56] -= 0.0833 * in(i + hipThreadIdx_x + 1, j, k + 8);
        buf0[57] -= 0.0833 * in(i + hipThreadIdx_x, j, k + 8);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 6; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[57 + rel] -= 0.166 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + _cg_idx1 + 1, k + 8);
            buf0[57 + rel] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + _cg_idx1 + 1, k + 8);
            buf0[58 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
            buf0[56 + rel] -= 0.0833 * in(i + hipThreadIdx_x, j + _cg_idx1 + 1, k + 8);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[63] -= 0.166 * in(i + hipThreadIdx_x, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + -1, j + 7, k + 8);
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x + 1, j + 7, k + 8);
        buf0[62] -= 0.0833 * in(i + hipThreadIdx_x, j + 7, k + 8);
      }
      {
        buf0[63] -= 0.0833 * in(i + hipThreadIdx_x, j + 8, k + 8);
      }
    }
    {
      long rel = 0;
      for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
      {
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
          {
            out(i + _cg_idx0, j + _cg_idx1, k + _cg_idx2) = buf0[rel];
          }
        }
      }
    }
  }
}
# 119 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
#undef in
#undef out
__global__ void poisson_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-poisson5.py-HIP-8x8x8-8x8" 1
{
  auto *binfo = out.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = binfo->adj[b][4];
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  long neighbor9 = binfo->adj[b][9];
  long neighbor10 = binfo->adj[b][10];
  long neighbor11 = binfo->adj[b][11];
  long neighbor12 = binfo->adj[b][12];
  long neighbor13 = b;
  long neighbor14 = binfo->adj[b][14];
  long neighbor15 = binfo->adj[b][15];
  long neighbor16 = binfo->adj[b][16];
  long neighbor17 = binfo->adj[b][17];
  long neighbor18 = binfo->adj[b][18];
  long neighbor19 = binfo->adj[b][19];
  long neighbor20 = binfo->adj[b][20];
  long neighbor21 = binfo->adj[b][21];
  long neighbor22 = binfo->adj[b][22];
  long neighbor23 = binfo->adj[b][23];
  long neighbor24 = binfo->adj[b][24];
  long neighbor25 = binfo->adj[b][25];
  long neighbor26 = binfo->adj[b][26];
  bElem buf0[8];
  {
    {
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 8; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [0, 0, 0]
            buf0[0 + rel] = 0;
          }
          _cg_rel2 += 1;
        }
      }
    }
    {
      bElem _cg_in000_vecbuf;
      bElem _cg_in_100_vecbuf;
      {
        // New offset [0, -1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor1 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor3 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, -1]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor5 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, -1]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor7 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor4 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, -1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] += 2.666 * _cg_in000_reg;
        buf0[1] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 0]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[0] -= 0.166 * _cg_in000_reg;
        buf0[1] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 0]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[0] -= 0.0833 * _cg_in000_reg;
      }
      {
        long _cg_rel2 = 0;
        for (long _cg_idx2 = 0; _cg_idx2 < 6; _cg_idx2 += 1)
        {
          long rel = _cg_rel2;
          {
            // New offset [-1, -1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor9 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor10 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, -1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor11 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] += 2.666 * _cg_in000_reg;
            buf0[2 + rel] -= 0.166 * _cg_in000_reg;
            buf0[0 + rel] -= 0.166 * _cg_in000_reg;
          }
          {
            // New offset [1, 0, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp0;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
              dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp0;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [-1, 1, 1]
            bElem _cg_in000_reg;
            {
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor15 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor12 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
              dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              _cg_vectmp2 = in.dat[neighbor16 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp3;
              _cg_vectmp3 = in.dat[neighbor13 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp4;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
              dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp4;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [0, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in000_reg = _cg_in000_vecbuf;
            }
            buf0[1 + rel] -= 0.166 * _cg_in000_reg;
            buf0[2 + rel] -= 0.0833 * _cg_in000_reg;
            buf0[0 + rel] -= 0.0833 * _cg_in000_reg;
          }
          {
            // New offset [1, 1, 1]
            bElem _cg_in000_reg;
            {
              _cg_in_100_vecbuf = _cg_in000_vecbuf;
              bElem _cg_vectmp0;
              _cg_vectmp0 = in.dat[neighbor17 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              bElem _cg_vectmp1;
              _cg_vectmp1 = in.dat[neighbor14 * in.step + 64 + (hipThreadIdx_x + rel * 64)];
              // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
              dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
              bElem _cg_vectmp2;
              // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
              dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
              _cg_in000_reg = _cg_vectmp2;
            }
            buf0[1 + rel] -= 0.0833 * _cg_in000_reg;
          }
          _cg_rel2 += 1;
        }
      }
      {
        // New offset [-1, -1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor9 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor10 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, -1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor11 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] += 2.666 * _cg_in000_reg;
        buf0[6] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 1, 7]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor15 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor12 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in_100_vecbuf
          dev_shl(_cg_in_100_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          _cg_vectmp2 = in.dat[neighbor16 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp3;
          _cg_vectmp3 = in.dat[neighbor13 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp3 ,_cg_vectmp2, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp3, _cg_vectmp2, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp4;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp4
          dev_shl(_cg_vectmp4, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp4;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
        buf0[6] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [1, 1, 7]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor17 * in.step + 448 + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor14 * in.step + 448 + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          bElem _cg_vectmp2;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp2
          dev_shl(_cg_vectmp2, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp2;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, -1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor19 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [-1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = in.dat[neighbor21 * in.step + hipThreadIdx_x];
          _cg_in000_vecbuf = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.166 * _cg_in000_reg;
      }
      {
        // New offset [1, 0, 8]
        bElem _cg_in000_reg;
        {
          _cg_in_100_vecbuf = _cg_in000_vecbuf;
          _cg_in000_vecbuf = in.dat[neighbor23 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_in_100_vecbuf ,_cg_in000_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_in_100_vecbuf, _cg_in000_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_in000_reg = _cg_vectmp0;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
      {
        // New offset [0, 1, 8]
        bElem _cg_in000_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = in.dat[neighbor25 * in.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = in.dat[neighbor22 * in.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_in000_vecbuf
          dev_shl(_cg_in000_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_in000_reg = _cg_in000_vecbuf;
        }
        buf0[7] -= 0.0833 * _cg_in000_reg;
      }
    }
    bElem *out_ref = &out.dat[neighbor13 * out.step];
    for (long sti = 0; sti < 8; ++sti)
    {
      out_ref[sti * 64 + hipThreadIdx_x] = buf0[sti];
    }
  }
}
# 141 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson.cu" 2

}
