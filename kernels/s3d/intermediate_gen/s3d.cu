#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
// #include "../out/laplacian-stencils.h"
#include "../../../gen/consts.h"
#include "./s3d.h"
#include <brick-hip.h>

__global__ void s3d_naive2(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem* dev_coeff) {
    const size_t radius = 2;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem temp = dev_coeff[0] * in[k][j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[k][j][i + a] + in[k][j + a][i] + in[k + a][j][i] +
            in[k][j][i - a] + in[k][j - a][i] + in[k - a][j][i]);
    }
    out[k][j][i] = temp;
}
__global__ void s3d_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bOut[b][k][j][i] = dev_coeff[0] * bIn[b][k][j][i];
    const size_t radius = 2;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][k][j][i] += dev_coeff[a] * (
            bIn[b][k][j][i + a] + bIn[b][k][j + a][i] + bIn[b][k + a][j][i] +
            bIn[b][k][j][i - a] + bIn[b][k][j - a][i] + bIn[b][k - a][j][i]
        );
    }
}
__global__ void s3d_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian2.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]
__global__ void s3d_codegen2(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem *dev_coeff) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian2.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef bIn
#undef bOut
__global__ void s3d_naive3(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem* dev_coeff) {
    const size_t radius = 3;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem temp = dev_coeff[0] * in[k][j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[k][j][i + a] + in[k][j + a][i] + in[k + a][j][i] +
            in[k][j][i - a] + in[k][j - a][i] + in[k - a][j][i]);
    }
    out[k][j][i] = temp;
}
__global__ void s3d_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bOut[b][k][j][i] = dev_coeff[0] * bIn[b][k][j][i];
    const size_t radius = 3;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][k][j][i] += dev_coeff[a] * (
            bIn[b][k][j][i + a] + bIn[b][k][j + a][i] + bIn[b][k + a][j][i] +
            bIn[b][k][j][i - a] + bIn[b][k][j - a][i] + bIn[b][k - a][j][i]
        );
    }
}
__global__ void s3d_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian3.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]
__global__ void s3d_codegen3(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem *dev_coeff) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian3.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef bIn
#undef bOut
__global__ void s3d_naive5(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem* dev_coeff) {
    const size_t radius = 5;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem temp = dev_coeff[0] * in[k][j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[k][j][i + a] + in[k][j + a][i] + in[k + a][j][i] +
            in[k][j][i - a] + in[k][j - a][i] + in[k - a][j][i]);
    }
    out[k][j][i] = temp;
}
__global__ void s3d_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bOut[b][k][j][i] = dev_coeff[0] * bIn[b][k][j][i];
    const size_t radius = 5;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][k][j][i] += dev_coeff[a] * (
            bIn[b][k][j][i + a] + bIn[b][k][j + a][i] + bIn[b][k + a][j][i] +
            bIn[b][k][j][i - a] + bIn[b][k][j - a][i] + bIn[b][k - a][j][i]
        );
    }
}
__global__ void s3d_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian5.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]
__global__ void s3d_codegen5(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem *dev_coeff) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s3d/intermediate_gen/laplacian5.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef bIn
#undef bOut
