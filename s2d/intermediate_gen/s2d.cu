#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
// #include "../out/laplacian-stencils.h"
#include "../../../gen/consts.h"
#include "./s2d.h"
#include <brick-hip.h>

__global__ void s2d_naive2(bElem (*in)[STRIDE0], bElem (*out)[STRIDE0], bElem* dev_coeff) {
    const size_t radius = 2;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    bElem temp = dev_coeff[0] * in[j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[j][i + a] + in[j + a][i] +
            in[j][i - a] + in[j - a][i]);
    }
    out[j][i] = temp;
}
__global__ void s2d_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    bOut[b][j][i] = dev_coeff[0] * bIn[b][j][i];
    const size_t radius = 2;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][j][i] += dev_coeff[a] * (
            bIn[b][j][i + a] + bIn[b][j + a][i] +
            bIn[b][j][i - a] + bIn[b][j - a][i]
        );
    }
}
__global__ void s2d_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s2d/intermediate_gen/laplacian2d2.py", VSVEC, (TILE1, TILE0), (FOLD), b);
}
#define bIn(a, b) arr_in[c][b]
#define bOut(a, b) arr_out[c][b]
__global__ void laplacian_codegen(bElem (*arr_in)[STRIDE0], bElem (*arr_out)[STRIDE0], bElem *dev_coeff) {
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s2d/intermediate_gen/laplacian2d2.py", VSVEC, (TILE1, VECSIZE), ("j", "i"), (1, VECSIZE));
}
#undef bIn
#undef bOut
