#include "hip/hip_runtime.h"
# 1 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s2d/intermediate_gen/s2d.cu"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
// #include "../out/laplacian-stencils.h"
#include "../../../gen/consts.h"
#include "./s2d.h"
#include <brick-hip.h>

__global__ void s2d_naive2(bElem (*in)[STRIDE0], bElem (*out)[STRIDE0], bElem* dev_coeff) {
    const size_t radius = 2;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    bElem temp = dev_coeff[0] * in[j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[j][i + a] + in[j + a][i] +
            in[j][i - a] + in[j - a][i]);
    }
    out[j][i] = temp;
}
__global__ void s2d_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    bOut[b][j][i] = dev_coeff[0] * bIn[b][j][i];
    const size_t radius = 2;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][j][i] += dev_coeff[a] * (
            bIn[b][j][i + a] + bIn[b][j + a][i] +
            bIn[b][j][i - a] + bIn[b][j - a][i]
        );
    }
}
__global__ void s2d_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
# 1 "VSBrick-laplacian2d2.py-HIP-8x8-8x8" 1
{
  auto *binfo = bOut.bInfo;
  long neighbor0 = binfo->adj[b][0];
  long neighbor1 = binfo->adj[b][1];
  long neighbor2 = binfo->adj[b][2];
  long neighbor3 = binfo->adj[b][3];
  long neighbor4 = b;
  long neighbor5 = binfo->adj[b][5];
  long neighbor6 = binfo->adj[b][6];
  long neighbor7 = binfo->adj[b][7];
  long neighbor8 = binfo->adj[b][8];
  bElem buf0[1];
  {
    {
      {
        // New offset [0, 0]
        buf0[0] = 0;
      }
    }
    {
      bElem _cg_bIn00_vecbuf;
      bElem _cg_bIn_10_vecbuf;
      {
        // New offset [0, -2]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = bIn.dat[neighbor4 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = bIn.dat[neighbor1 * bIn.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 6 -> _cg_bIn00_vecbuf
          dev_shl(_cg_bIn00_vecbuf, _cg_vectmp1, _cg_vectmp0, 16, 64, hipThreadIdx_x);
          _cg_bIn00_reg = _cg_bIn00_vecbuf;
        }
        buf0[0] += dev_coeff[2] * _cg_bIn00_reg;
      }
      {
        // New offset [0, -1]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = bIn.dat[neighbor4 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = bIn.dat[neighbor1 * bIn.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 7 -> _cg_bIn00_vecbuf
          dev_shl(_cg_bIn00_vecbuf, _cg_vectmp1, _cg_vectmp0, 8, 64, hipThreadIdx_x);
          _cg_bIn00_reg = _cg_bIn00_vecbuf;
        }
        buf0[0] += dev_coeff[1] * _cg_bIn00_reg;
      }
      {
        // New offset [-2, 0]
        bElem _cg_bIn00_reg;
        {
          _cg_bIn_10_vecbuf = bIn.dat[neighbor3 * bIn.step + hipThreadIdx_x];
          _cg_bIn00_vecbuf = bIn.dat[neighbor4 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_bIn_10_vecbuf ,_cg_bIn00_vecbuf, 6 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_bIn_10_vecbuf, _cg_bIn00_vecbuf, 2, 8, hipThreadIdx_x & 7);
          _cg_bIn00_reg = _cg_vectmp0;
        }
        buf0[0] += dev_coeff[2] * _cg_bIn00_reg;
      }
      {
        // New offset [-1, 0]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          // merge0 _cg_bIn_10_vecbuf ,_cg_bIn00_vecbuf, 7 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_bIn_10_vecbuf, _cg_bIn00_vecbuf, 1, 8, hipThreadIdx_x & 7);
          _cg_bIn00_reg = _cg_vectmp0;
        }
        buf0[0] += dev_coeff[1] * _cg_bIn00_reg;
      }
      {
        // New offset [0, 0]
        bElem _cg_bIn00_reg;
        {
          _cg_bIn00_reg = _cg_bIn00_vecbuf;
        }
        buf0[0] += dev_coeff[0] * _cg_bIn00_reg;
      }
      {
        // New offset [1, 0]
        bElem _cg_bIn00_reg;
        {
          _cg_bIn_10_vecbuf = _cg_bIn00_vecbuf;
          _cg_bIn00_vecbuf = bIn.dat[neighbor5 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp0;
          // merge0 _cg_bIn_10_vecbuf ,_cg_bIn00_vecbuf, 1 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_bIn_10_vecbuf, _cg_bIn00_vecbuf, 7, 8, hipThreadIdx_x & 7);
          _cg_bIn00_reg = _cg_vectmp0;
        }
        buf0[0] += dev_coeff[1] * _cg_bIn00_reg;
      }
      {
        // New offset [2, 0]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          // merge0 _cg_bIn_10_vecbuf ,_cg_bIn00_vecbuf, 2 -> _cg_vectmp0
          dev_shl(_cg_vectmp0, _cg_bIn_10_vecbuf, _cg_bIn00_vecbuf, 6, 8, hipThreadIdx_x & 7);
          _cg_bIn00_reg = _cg_vectmp0;
        }
        buf0[0] += dev_coeff[2] * _cg_bIn00_reg;
      }
      {
        // New offset [0, 1]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = bIn.dat[neighbor7 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = bIn.dat[neighbor4 * bIn.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 1 -> _cg_bIn00_vecbuf
          dev_shl(_cg_bIn00_vecbuf, _cg_vectmp1, _cg_vectmp0, 56, 64, hipThreadIdx_x);
          _cg_bIn00_reg = _cg_bIn00_vecbuf;
        }
        buf0[0] += dev_coeff[1] * _cg_bIn00_reg;
      }
      {
        // New offset [0, 2]
        bElem _cg_bIn00_reg;
        {
          bElem _cg_vectmp0;
          _cg_vectmp0 = bIn.dat[neighbor7 * bIn.step + hipThreadIdx_x];
          bElem _cg_vectmp1;
          _cg_vectmp1 = bIn.dat[neighbor4 * bIn.step + hipThreadIdx_x];
          // merge1 _cg_vectmp1 ,_cg_vectmp0, 2 -> _cg_bIn00_vecbuf
          dev_shl(_cg_bIn00_vecbuf, _cg_vectmp1, _cg_vectmp0, 48, 64, hipThreadIdx_x);
          _cg_bIn00_reg = _cg_bIn00_vecbuf;
        }
        buf0[0] += dev_coeff[2] * _cg_bIn00_reg;
      }
    }
    bElem *bOut_ref = &bOut.dat[neighbor4 * bOut.step];
    for (long sti = 0; sti < 1; ++sti)
    {
      bOut_ref[sti * 64 + hipThreadIdx_x] = buf0[sti];
    }
  }
}
# 38 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s2d/intermediate_gen/s2d.cu" 2

}
#define bIn(a, b) arr_in[c][b]
#define bOut(a, b) arr_out[c][b]
__global__ void laplacian_codegen(bElem (*arr_in)[STRIDE0], bElem (*arr_out)[STRIDE0], bElem *dev_coeff) {
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
# 1 "VSTile-laplacian2d2.py-HIP-8x64" 1
{
  bElem buf0[8];
  {
    {
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[0 + rel] = 0;
          }
          _cg_rel1 += 1;
        }
      }
    }
    {
      {
        buf0[0] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + -2);
      }
      {
        buf0[0] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + -1);
        buf0[1] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + -1);
      }
      {
        buf0[0] += dev_coeff[0] * bIn(i + hipThreadIdx_x, j);
        buf0[0] += dev_coeff[1] * bIn(i + hipThreadIdx_x + 1, j);
        buf0[0] += dev_coeff[1] * bIn(i + hipThreadIdx_x + -1, j);
        buf0[0] += dev_coeff[2] * bIn(i + hipThreadIdx_x + 2, j);
        buf0[0] += dev_coeff[2] * bIn(i + hipThreadIdx_x + -2, j);
        buf0[1] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j);
        buf0[2] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j);
      }
      {
        buf0[1] += dev_coeff[0] * bIn(i + hipThreadIdx_x, j + 1);
        buf0[1] += dev_coeff[1] * bIn(i + hipThreadIdx_x + 1, j + 1);
        buf0[1] += dev_coeff[1] * bIn(i + hipThreadIdx_x + -1, j + 1);
        buf0[1] += dev_coeff[2] * bIn(i + hipThreadIdx_x + 2, j + 1);
        buf0[1] += dev_coeff[2] * bIn(i + hipThreadIdx_x + -2, j + 1);
        buf0[0] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 1);
        buf0[2] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 1);
        buf0[3] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + 1);
      }
      {
        long _cg_rel1 = 0;
        for (long _cg_idx1 = 0; _cg_idx1 < 4; _cg_idx1 += 1)
        {
          long rel = _cg_rel1;
          {
            buf0[2 + rel] += dev_coeff[0] * bIn(i + hipThreadIdx_x, j + _cg_idx1 + 2);
            buf0[2 + rel] += dev_coeff[1] * bIn(i + hipThreadIdx_x + 1, j + _cg_idx1 + 2);
            buf0[2 + rel] += dev_coeff[1] * bIn(i + hipThreadIdx_x + -1, j + _cg_idx1 + 2);
            buf0[2 + rel] += dev_coeff[2] * bIn(i + hipThreadIdx_x + 2, j + _cg_idx1 + 2);
            buf0[2 + rel] += dev_coeff[2] * bIn(i + hipThreadIdx_x + -2, j + _cg_idx1 + 2);
            buf0[1 + rel] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + _cg_idx1 + 2);
            buf0[3 + rel] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + _cg_idx1 + 2);
            buf0[0 + rel] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + _cg_idx1 + 2);
            buf0[4 + rel] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + _cg_idx1 + 2);
          }
          _cg_rel1 += 1;
        }
      }
      {
        buf0[6] += dev_coeff[0] * bIn(i + hipThreadIdx_x, j + 6);
        buf0[6] += dev_coeff[1] * bIn(i + hipThreadIdx_x + 1, j + 6);
        buf0[6] += dev_coeff[1] * bIn(i + hipThreadIdx_x + -1, j + 6);
        buf0[6] += dev_coeff[2] * bIn(i + hipThreadIdx_x + 2, j + 6);
        buf0[6] += dev_coeff[2] * bIn(i + hipThreadIdx_x + -2, j + 6);
        buf0[5] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 6);
        buf0[7] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 6);
        buf0[4] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + 6);
      }
      {
        buf0[7] += dev_coeff[0] * bIn(i + hipThreadIdx_x, j + 7);
        buf0[7] += dev_coeff[1] * bIn(i + hipThreadIdx_x + 1, j + 7);
        buf0[7] += dev_coeff[1] * bIn(i + hipThreadIdx_x + -1, j + 7);
        buf0[7] += dev_coeff[2] * bIn(i + hipThreadIdx_x + 2, j + 7);
        buf0[7] += dev_coeff[2] * bIn(i + hipThreadIdx_x + -2, j + 7);
        buf0[6] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 7);
        buf0[5] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + 7);
      }
      {
        buf0[7] += dev_coeff[1] * bIn(i + hipThreadIdx_x, j + 8);
        buf0[6] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + 8);
      }
      {
        buf0[7] += dev_coeff[2] * bIn(i + hipThreadIdx_x, j + 9);
      }
    }
    {
      long rel = 0;
      for (long _cg_idx1 = 0; _cg_idx1 < 8; _cg_idx1 += 1)
      {
        for (long _cg_idx0 = hipThreadIdx_x; _cg_idx0 < 64; _cg_idx0 += 64, ++rel)
        {
          bOut(i + _cg_idx0, j + _cg_idx1) = buf0[rel];
        }
      }
    }
  }
}
# 45 "/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/s2d/intermediate_gen/s2d.cu" 2

}
#undef bIn
#undef bOut
