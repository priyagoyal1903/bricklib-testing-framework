#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void poisson_naive(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
// $END naive

// $START codegen
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef in
#undef out
// $END codegen

// $START naive-bricks
__global__ void poisson_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;

    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
// $END naive-bricks

// $START codegen-bricks
__global__ void poisson_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks
