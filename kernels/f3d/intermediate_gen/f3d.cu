#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./f3d.h"
#include <brick-hip.h>

__global__ void f3d_naive1(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    const int radius = 1;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem base = in[i][j][k] * c[i][j][k];
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {		
		base += (in[i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
            }
        }
    }
    out[i][j][k] = base;
}
__global__ void f3d_naive_bricks1(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    const int radius = 1;
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem base = bIn[b][i][j][k] * c[i][j][k];
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {
                base += (bIn[b][i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
	    }
        }
    }
    bOut[b][i][j][k] = base;
}
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]
__global__ void f3d_codegen1(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/f3d/intermediate_gen/f3d1.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef bIn
#undef bOut
__global__ void f3d_codegen_bricks1(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/f3d/intermediate_gen/f3d1.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void f3d_naive2(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    const int radius = 2;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem base = in[i][j][k] * c[i][j][k];
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {		
		base += (in[i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
            }
        }
    }
    out[i][j][k] = base;
}
__global__ void f3d_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    const int radius = 2;
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem base = bIn[b][i][j][k] * c[i][j][k];
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {
                base += (bIn[b][i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
	    }
        }
    }
    bOut[b][i][j][k] = base;
}
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]
__global__ void f3d_codegen2(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/f3d/intermediate_gen/f3d2.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef bIn
#undef bOut
__global__ void f3d_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/f3d/intermediate_gen/f3d2.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
