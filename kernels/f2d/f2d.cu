#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"


// $START naive
__global__ void f2d_naive(bElem (*in)[STRIDE0], bElem (*out)[STRIDE0], bElem (*c)[8]) {
    const size_t radius = $SIZE;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;

    bElem base = 0;
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            base += (in[i + i_diff][j + j_diff] * c[i_diff + radius][j_diff + radius]);
        }
    }
    out[i][j] = base;
}
// $END naive

// $START naive-bricks
__global__ void f2d_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8]) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;

    bElem base = 0;
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            base += (bIn[b][i + i_diff][j + j_diff] * c[i_diff + radius][j_diff + radius]);
        }
    }
    bOut[b][i][j] = base;
}
// $END naive-brick

// $START codegen
#define bIn(a, b) arr_in[b][a]
#define bOut(a, b) arr_out[b][a]

__global__ void f2d_codegen(bElem (*arr_in)[STRIDE0], bElem (*arr_out)[STRIDE0], bElem (*c)[8]) {
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("$PYTHON", VSVEC, (TILE1, VECSIZE), ("j", "i"), (1, VECSIZE));
}

#undef bIn
#undef bOut
// $END codegen

// $START codegen-bricks
__global__ void f32_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8]) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    brick("$PYTHON", VSVEC, (TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks
