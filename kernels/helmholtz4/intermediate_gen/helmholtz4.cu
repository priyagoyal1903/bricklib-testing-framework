#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./helmholtz4.h"
#include <brick-hip.h>

__global__ void helmholtz4_naive2(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], 
  bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], 
  bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[k][j][i] * (15.0 * (x[k][j][i - 1] - x[k][j][i]) - 
                (x[k][j][i - 1] - x[k][j][i + 1])) + 
            beta_i[k][j][i + 1] * (15.0 * (x[k][j][i + 1] - x[k][j][i]) - 
                (x[k][j][i + 2] - x[k][j][i - 1])) +
            beta_j[k][j][i] * (15.0 * (x[k][j - 1][i] - x[k][j][i]) - 
                (x[k][j - 1][i] - x[k][j + 1][i])) +
            beta_j[k][j + 1][i] * (15.0 * (x[k][j + 1][i] - x[k][j][i]) -
                (x[k][j + 2][i] - x[k][j - 1][i])) +
            beta_k[k][j][i] * (15.0 * (x[k - 1][j][i] - x[k][j][i]) -
                (x[k - 2][j][i] - x[k + 1][j][i])) +
            beta_k[k + 1][j][i] * (15.0 * (x[k + 1][j][i] - x[k][j][i]) -
                (x[k + 2][j][i] - x[k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[k][j + 1][i] - beta_i[k][j - 1][i]) *
                (x[k][j + 1][i - 1] - x[k][j + 1][i] -
                 x[k][j - 1][i - 1] + x[k][j - 1][i]) +
            (beta_i[k + 1][j][i] - beta_i[k - 1][j][i]) * 
                (x[k + 1][j][i - 1] - x[k + 1][j][i] -
                 x[k - 1][j][i - 1] + x[k - 1][j][i]) +
            (beta_j[k][j][i + 1] - beta_j[k][j][i - 1]) *
                (x[k][j - 1][i + 1] - x[k][j][i + 1] -
                 x[k][j - 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j][i] - beta_j[k - 1][j][i]) *
                (x[k + 1][j - 1][i] - x[k + 1][j][i] -
                 x[k - 1][j - 1][i] + x[k - 1][j][i]) +
            (beta_k[k][j][i + 1] - beta_k[k][j][i - 1]) *
                (x[k - 1][j][i + 1] - x[k][j][i + 1] -
                 x[k - 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k][j + 1][i] - beta_k[k][j - 1][i]) *
                (x[k - 1][j + 1][i] - x[k][j + 1][i] -
                 x[k - 1][j - 1][i] + x[k][j - 1][i]) +
            (beta_i[k][j + 1][i + 1] - beta_i[k][j - 1][i + 1]) *
                (x[k][j + 1][i + 1] - x[k][j + 1][i] -
                 x[k][j - 1][i + 1] + x[k][j - 1][i]) + 
            (beta_i[k + 1][j][i + 1] - beta_i[k - 1][j][i + 1]) *
                (x[k + 1][j][i + 1] - x[k + 1][j][i] - 
                 x[k - 1][j][i + 1] + x[k - 1][j][i]) +
            (beta_j[k][j + 1][i + 1] - beta_j[k][j + 1][i - 1]) *
                (x[k][j + 1][i + 1] - x[k][j][i + 1] -
                 x[k][j + 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j + 1][i] - beta_j[k - 1][j + 1][i]) *
                (x[k + 1][j + 1][i] - x[k + 1][j][i] -
                 x[k - 1][j + 1][i] + x[k - 1][j][i]) +
            (beta_k[k + 1][j][i + 1] - beta_k[k + 1][j][i - 1]) *
                (x[k + 1][j][i + 1] - x[k][j][i + 1] -
                 x[k + 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k + 1][j + 1][i] - beta_k[k + 1][j - 1][i]) *
                (x[k + 1][j + 1][i] - x[k][j + 1][i] -
                 x[k + 1][j - 1][i] + x[k][j - 1][i])
        ));
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz4_codegen2(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], 
  bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], 
  bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz42.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz4_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[b][k][j][i] * (15.0 * (x[b][k][j][i - 1] - x[b][k][j][i]) - 
                (x[b][k][j][i - 1] - x[b][k][j][i + 1])) + 
            beta_i[b][k][j][i + 1] * (15.0 * (x[b][k][j][i + 1] - x[b][k][j][i]) - 
                (x[b][k][j][i + 2] - x[b][k][j][i - 1])) +
            beta_j[b][k][j][i] * (15.0 * (x[b][k][j - 1][i] - x[b][k][j][i]) - 
                (x[b][k][j - 1][i] - x[b][k][j + 1][i])) +
            beta_j[b][k][j + 1][i] * (15.0 * (x[b][k][j + 1][i] - x[b][k][j][i]) -
                (x[b][k][j + 2][i] - x[b][k][j - 1][i])) +
            beta_k[b][k][j][i] * (15.0 * (x[b][k - 1][j][i] - x[b][k][j][i]) -
                (x[b][k - 2][j][i] - x[b][k + 1][j][i])) +
            beta_k[b][k + 1][j][i] * (15.0 * (x[b][k + 1][j][i] - x[b][k][j][i]) -
                (x[b][k + 2][j][i] - x[b][k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[b][k][j + 1][i] - beta_i[b][k][j - 1][i]) *
                (x[b][k][j + 1][i - 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i - 1] + x[b][k][j - 1][i]) +
            (beta_i[b][k + 1][j][i] - beta_i[b][k - 1][j][i]) * 
                (x[b][k + 1][j][i - 1] - x[b][k + 1][j][i] -
                 x[b][k - 1][j][i - 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j][i + 1] - beta_j[b][k][j][i - 1]) *
                (x[b][k][j - 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j - 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j][i] - beta_j[b][k - 1][j][i]) *
                (x[b][k + 1][j - 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j - 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k][j][i + 1] - beta_k[b][k][j][i - 1]) *
                (x[b][k - 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k - 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k][j + 1][i] - beta_k[b][k][j - 1][i]) *
                (x[b][k - 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k - 1][j - 1][i] + x[b][k][j - 1][i]) +
            (beta_i[b][k][j + 1][i + 1] - beta_i[b][k][j - 1][i + 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i + 1] + x[b][k][j - 1][i]) + 
            (beta_i[b][k + 1][j][i + 1] - beta_i[b][k - 1][j][i + 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k + 1][j][i] - 
                 x[b][k - 1][j][i + 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j + 1][i + 1] - beta_j[b][k][j + 1][i - 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j + 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j + 1][i] - beta_j[b][k - 1][j + 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j + 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k + 1][j][i + 1] - beta_k[b][k + 1][j][i - 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k + 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k + 1][j + 1][i] - beta_k[b][k + 1][j - 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k + 1][j - 1][i] + x[b][k][j - 1][i])
        ));
} 
__global__ void helmholtz4_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz42.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void helmholtz4_naive3(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], 
  bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], 
  bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[k][j][i] * (15.0 * (x[k][j][i - 1] - x[k][j][i]) - 
                (x[k][j][i - 1] - x[k][j][i + 1])) + 
            beta_i[k][j][i + 1] * (15.0 * (x[k][j][i + 1] - x[k][j][i]) - 
                (x[k][j][i + 2] - x[k][j][i - 1])) +
            beta_j[k][j][i] * (15.0 * (x[k][j - 1][i] - x[k][j][i]) - 
                (x[k][j - 1][i] - x[k][j + 1][i])) +
            beta_j[k][j + 1][i] * (15.0 * (x[k][j + 1][i] - x[k][j][i]) -
                (x[k][j + 2][i] - x[k][j - 1][i])) +
            beta_k[k][j][i] * (15.0 * (x[k - 1][j][i] - x[k][j][i]) -
                (x[k - 2][j][i] - x[k + 1][j][i])) +
            beta_k[k + 1][j][i] * (15.0 * (x[k + 1][j][i] - x[k][j][i]) -
                (x[k + 2][j][i] - x[k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[k][j + 1][i] - beta_i[k][j - 1][i]) *
                (x[k][j + 1][i - 1] - x[k][j + 1][i] -
                 x[k][j - 1][i - 1] + x[k][j - 1][i]) +
            (beta_i[k + 1][j][i] - beta_i[k - 1][j][i]) * 
                (x[k + 1][j][i - 1] - x[k + 1][j][i] -
                 x[k - 1][j][i - 1] + x[k - 1][j][i]) +
            (beta_j[k][j][i + 1] - beta_j[k][j][i - 1]) *
                (x[k][j - 1][i + 1] - x[k][j][i + 1] -
                 x[k][j - 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j][i] - beta_j[k - 1][j][i]) *
                (x[k + 1][j - 1][i] - x[k + 1][j][i] -
                 x[k - 1][j - 1][i] + x[k - 1][j][i]) +
            (beta_k[k][j][i + 1] - beta_k[k][j][i - 1]) *
                (x[k - 1][j][i + 1] - x[k][j][i + 1] -
                 x[k - 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k][j + 1][i] - beta_k[k][j - 1][i]) *
                (x[k - 1][j + 1][i] - x[k][j + 1][i] -
                 x[k - 1][j - 1][i] + x[k][j - 1][i]) +
            (beta_i[k][j + 1][i + 1] - beta_i[k][j - 1][i + 1]) *
                (x[k][j + 1][i + 1] - x[k][j + 1][i] -
                 x[k][j - 1][i + 1] + x[k][j - 1][i]) + 
            (beta_i[k + 1][j][i + 1] - beta_i[k - 1][j][i + 1]) *
                (x[k + 1][j][i + 1] - x[k + 1][j][i] - 
                 x[k - 1][j][i + 1] + x[k - 1][j][i]) +
            (beta_j[k][j + 1][i + 1] - beta_j[k][j + 1][i - 1]) *
                (x[k][j + 1][i + 1] - x[k][j][i + 1] -
                 x[k][j + 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j + 1][i] - beta_j[k - 1][j + 1][i]) *
                (x[k + 1][j + 1][i] - x[k + 1][j][i] -
                 x[k - 1][j + 1][i] + x[k - 1][j][i]) +
            (beta_k[k + 1][j][i + 1] - beta_k[k + 1][j][i - 1]) *
                (x[k + 1][j][i + 1] - x[k][j][i + 1] -
                 x[k + 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k + 1][j + 1][i] - beta_k[k + 1][j - 1][i]) *
                (x[k + 1][j + 1][i] - x[k][j + 1][i] -
                 x[k + 1][j - 1][i] + x[k][j - 1][i])
        ));
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz4_codegen3(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], 
  bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], 
  bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz43.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz4_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[b][k][j][i] * (15.0 * (x[b][k][j][i - 1] - x[b][k][j][i]) - 
                (x[b][k][j][i - 1] - x[b][k][j][i + 1])) + 
            beta_i[b][k][j][i + 1] * (15.0 * (x[b][k][j][i + 1] - x[b][k][j][i]) - 
                (x[b][k][j][i + 2] - x[b][k][j][i - 1])) +
            beta_j[b][k][j][i] * (15.0 * (x[b][k][j - 1][i] - x[b][k][j][i]) - 
                (x[b][k][j - 1][i] - x[b][k][j + 1][i])) +
            beta_j[b][k][j + 1][i] * (15.0 * (x[b][k][j + 1][i] - x[b][k][j][i]) -
                (x[b][k][j + 2][i] - x[b][k][j - 1][i])) +
            beta_k[b][k][j][i] * (15.0 * (x[b][k - 1][j][i] - x[b][k][j][i]) -
                (x[b][k - 2][j][i] - x[b][k + 1][j][i])) +
            beta_k[b][k + 1][j][i] * (15.0 * (x[b][k + 1][j][i] - x[b][k][j][i]) -
                (x[b][k + 2][j][i] - x[b][k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[b][k][j + 1][i] - beta_i[b][k][j - 1][i]) *
                (x[b][k][j + 1][i - 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i - 1] + x[b][k][j - 1][i]) +
            (beta_i[b][k + 1][j][i] - beta_i[b][k - 1][j][i]) * 
                (x[b][k + 1][j][i - 1] - x[b][k + 1][j][i] -
                 x[b][k - 1][j][i - 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j][i + 1] - beta_j[b][k][j][i - 1]) *
                (x[b][k][j - 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j - 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j][i] - beta_j[b][k - 1][j][i]) *
                (x[b][k + 1][j - 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j - 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k][j][i + 1] - beta_k[b][k][j][i - 1]) *
                (x[b][k - 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k - 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k][j + 1][i] - beta_k[b][k][j - 1][i]) *
                (x[b][k - 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k - 1][j - 1][i] + x[b][k][j - 1][i]) +
            (beta_i[b][k][j + 1][i + 1] - beta_i[b][k][j - 1][i + 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i + 1] + x[b][k][j - 1][i]) + 
            (beta_i[b][k + 1][j][i + 1] - beta_i[b][k - 1][j][i + 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k + 1][j][i] - 
                 x[b][k - 1][j][i + 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j + 1][i + 1] - beta_j[b][k][j + 1][i - 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j + 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j + 1][i] - beta_j[b][k - 1][j + 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j + 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k + 1][j][i + 1] - beta_k[b][k + 1][j][i - 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k + 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k + 1][j + 1][i] - beta_k[b][k + 1][j - 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k + 1][j - 1][i] + x[b][k][j - 1][i])
        ));
} 
__global__ void helmholtz4_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz43.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void helmholtz4_naive5(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], 
  bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], 
  bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[k][j][i] * (15.0 * (x[k][j][i - 1] - x[k][j][i]) - 
                (x[k][j][i - 1] - x[k][j][i + 1])) + 
            beta_i[k][j][i + 1] * (15.0 * (x[k][j][i + 1] - x[k][j][i]) - 
                (x[k][j][i + 2] - x[k][j][i - 1])) +
            beta_j[k][j][i] * (15.0 * (x[k][j - 1][i] - x[k][j][i]) - 
                (x[k][j - 1][i] - x[k][j + 1][i])) +
            beta_j[k][j + 1][i] * (15.0 * (x[k][j + 1][i] - x[k][j][i]) -
                (x[k][j + 2][i] - x[k][j - 1][i])) +
            beta_k[k][j][i] * (15.0 * (x[k - 1][j][i] - x[k][j][i]) -
                (x[k - 2][j][i] - x[k + 1][j][i])) +
            beta_k[k + 1][j][i] * (15.0 * (x[k + 1][j][i] - x[k][j][i]) -
                (x[k + 2][j][i] - x[k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[k][j + 1][i] - beta_i[k][j - 1][i]) *
                (x[k][j + 1][i - 1] - x[k][j + 1][i] -
                 x[k][j - 1][i - 1] + x[k][j - 1][i]) +
            (beta_i[k + 1][j][i] - beta_i[k - 1][j][i]) * 
                (x[k + 1][j][i - 1] - x[k + 1][j][i] -
                 x[k - 1][j][i - 1] + x[k - 1][j][i]) +
            (beta_j[k][j][i + 1] - beta_j[k][j][i - 1]) *
                (x[k][j - 1][i + 1] - x[k][j][i + 1] -
                 x[k][j - 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j][i] - beta_j[k - 1][j][i]) *
                (x[k + 1][j - 1][i] - x[k + 1][j][i] -
                 x[k - 1][j - 1][i] + x[k - 1][j][i]) +
            (beta_k[k][j][i + 1] - beta_k[k][j][i - 1]) *
                (x[k - 1][j][i + 1] - x[k][j][i + 1] -
                 x[k - 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k][j + 1][i] - beta_k[k][j - 1][i]) *
                (x[k - 1][j + 1][i] - x[k][j + 1][i] -
                 x[k - 1][j - 1][i] + x[k][j - 1][i]) +
            (beta_i[k][j + 1][i + 1] - beta_i[k][j - 1][i + 1]) *
                (x[k][j + 1][i + 1] - x[k][j + 1][i] -
                 x[k][j - 1][i + 1] + x[k][j - 1][i]) + 
            (beta_i[k + 1][j][i + 1] - beta_i[k - 1][j][i + 1]) *
                (x[k + 1][j][i + 1] - x[k + 1][j][i] - 
                 x[k - 1][j][i + 1] + x[k - 1][j][i]) +
            (beta_j[k][j + 1][i + 1] - beta_j[k][j + 1][i - 1]) *
                (x[k][j + 1][i + 1] - x[k][j][i + 1] -
                 x[k][j + 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j + 1][i] - beta_j[k - 1][j + 1][i]) *
                (x[k + 1][j + 1][i] - x[k + 1][j][i] -
                 x[k - 1][j + 1][i] + x[k - 1][j][i]) +
            (beta_k[k + 1][j][i + 1] - beta_k[k + 1][j][i - 1]) *
                (x[k + 1][j][i + 1] - x[k][j][i + 1] -
                 x[k + 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k + 1][j + 1][i] - beta_k[k + 1][j - 1][i]) *
                (x[k + 1][j + 1][i] - x[k][j + 1][i] -
                 x[k + 1][j - 1][i] + x[k][j - 1][i])
        ));
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz4_codegen5(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], 
  bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], 
  bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz45.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz4_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[b][k][j][i] * (15.0 * (x[b][k][j][i - 1] - x[b][k][j][i]) - 
                (x[b][k][j][i - 1] - x[b][k][j][i + 1])) + 
            beta_i[b][k][j][i + 1] * (15.0 * (x[b][k][j][i + 1] - x[b][k][j][i]) - 
                (x[b][k][j][i + 2] - x[b][k][j][i - 1])) +
            beta_j[b][k][j][i] * (15.0 * (x[b][k][j - 1][i] - x[b][k][j][i]) - 
                (x[b][k][j - 1][i] - x[b][k][j + 1][i])) +
            beta_j[b][k][j + 1][i] * (15.0 * (x[b][k][j + 1][i] - x[b][k][j][i]) -
                (x[b][k][j + 2][i] - x[b][k][j - 1][i])) +
            beta_k[b][k][j][i] * (15.0 * (x[b][k - 1][j][i] - x[b][k][j][i]) -
                (x[b][k - 2][j][i] - x[b][k + 1][j][i])) +
            beta_k[b][k + 1][j][i] * (15.0 * (x[b][k + 1][j][i] - x[b][k][j][i]) -
                (x[b][k + 2][j][i] - x[b][k - 1][j][i]))) +
        0.25 * 0.0833 * 
            ((beta_i[b][k][j + 1][i] - beta_i[b][k][j - 1][i]) *
                (x[b][k][j + 1][i - 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i - 1] + x[b][k][j - 1][i]) +
            (beta_i[b][k + 1][j][i] - beta_i[b][k - 1][j][i]) * 
                (x[b][k + 1][j][i - 1] - x[b][k + 1][j][i] -
                 x[b][k - 1][j][i - 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j][i + 1] - beta_j[b][k][j][i - 1]) *
                (x[b][k][j - 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j - 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j][i] - beta_j[b][k - 1][j][i]) *
                (x[b][k + 1][j - 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j - 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k][j][i + 1] - beta_k[b][k][j][i - 1]) *
                (x[b][k - 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k - 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k][j + 1][i] - beta_k[b][k][j - 1][i]) *
                (x[b][k - 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k - 1][j - 1][i] + x[b][k][j - 1][i]) +
            (beta_i[b][k][j + 1][i + 1] - beta_i[b][k][j - 1][i + 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i + 1] + x[b][k][j - 1][i]) + 
            (beta_i[b][k + 1][j][i + 1] - beta_i[b][k - 1][j][i + 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k + 1][j][i] - 
                 x[b][k - 1][j][i + 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j + 1][i + 1] - beta_j[b][k][j + 1][i - 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j + 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j + 1][i] - beta_j[b][k - 1][j + 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j + 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k + 1][j][i + 1] - beta_k[b][k + 1][j][i - 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k + 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k + 1][j + 1][i] - beta_k[b][k + 1][j - 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k + 1][j - 1][i] + x[b][k][j - 1][i])
        ));
} 
__global__ void helmholtz4_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz4/intermediate_gen/helmholtz45.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
