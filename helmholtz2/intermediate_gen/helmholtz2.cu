#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./helmholtz2.h"
#include <brick-hip.h>

__global__ void helmholtz2_naive2(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen2(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz22.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz22.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void helmholtz2_naive3(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen3(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz23.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz23.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void helmholtz2_naive5(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i] - 
        c2 * h2inv * (
            beta_i[k][j][i + 1] * (x[k][j][i + 1] - x[k][j][i]) + 
            beta_i[k][j][i]     * (x[k][j][i - 1] - x[k][j][i]) +
            beta_j[k][j + 1][i] * (x[k][j + 1][i] - x[k][j][i]) +
            beta_j[k][j - 1][i] * (x[k][j - 1][i] - x[k][j][i]) +
            beta_k[k + 1][j][i] * (x[k + 1][j][i] - x[k][j][i]) +
            beta_k[k - 1][j][i] * (x[k - 1][j][i] - x[k][j][i])
        );
}
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]
__global__ void helmholtz2_codegen5(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz25.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
__global__ void helmholtz2_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];
    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i] - 
        c2 * h2inv * (
            beta_i[b][k][j][i + 1] * (x[b][k][j][i + 1] - x[b][k][j][i]) + 
            beta_i[b][k][j][i]     * (x[b][k][j][i - 1] - x[b][k][j][i]) +
            beta_j[b][k][j + 1][i] * (x[b][k][j + 1][i] - x[b][k][j][i]) +
            beta_j[b][k][j - 1][i] * (x[b][k][j - 1][i] - x[b][k][j][i]) +
            beta_k[b][k + 1][j][i] * (x[b][k + 1][j][i] - x[b][k][j][i]) +
            beta_k[b][k - 1][j][i] * (x[b][k - 1][j][i] - x[b][k][j][i])
        );
}
__global__ void helmholtz2_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha, BType beta_i, BType beta_j, BType beta_k, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/helmholtz2/intermediate_gen/helmholtz25.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
