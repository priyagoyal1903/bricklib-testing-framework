#include "hip/hip_runtime.h"

#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// #include "../out/laplacian-stencils.h"

// $START naive
__global__ void laplacian_naive(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem* dev_coeff) {
    const size_t radius = $SIZE;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    bElem temp = dev_coeff[0] * in[k][j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[k][j][i + a] + in[k][j + a][i] + in[k + a][j][i] +
            in[k][j][i - a] + in[k][j - a][i] + in[k - a][j][i]);
    }
    out[k][j][i] = temp;
}
// $END naive

// $START naive-bricks
__global__ void laplacian_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bOut[b][k][j][i] = dev_coeff[0] * bIn[b][k][j][i];

    const size_t radius = $SIZE;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][k][j][i] += dev_coeff[a] * (
            bIn[b][k][j][i + a] + bIn[b][k][j + a][i] + bIn[b][k + a][j][i] +
            bIn[b][k][j][i - a] + bIn[b][k][j - a][i] + bIn[b][k - a][j][i]
        );
    }
}
// $END naive-bricks

// $START codegen-bricks
__global__ void laplacian_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks


// $START codegen
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]

__global__ void laplacian_codegen(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem *dev_coeff) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef bIn
#undef bOut
// $END codegen
