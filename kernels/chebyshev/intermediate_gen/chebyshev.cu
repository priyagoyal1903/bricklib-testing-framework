#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./chebyshev.h"
#include <brick-hip.h>

__global__ void chebyshev_naive1(bElem (*Ac)[STRIDE1][STRIDE0], bElem (*Ap)[STRIDE1][STRIDE0], bElem (*Dinv)[STRIDE1][STRIDE0], bElem (*RHS)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
  unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
  unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
  unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
  out[k][j][i] = 
    Ac[k][j][i] + c[0] * (Ac[k][j][i] + Ap[k][j][i]) +
    c[1] * Dinv[k][j][i] *
    (RHS[k][j][i] +
    (Ac[k][j][i] + 
    c[2] * 
      (
        0.03 * (Ac[k - 1][j - 1][i - 1] + Ac[k - 1][j - 1][i + 1] +
                Ac[k - 1][j + 1][i - 1] + Ac[k - 1][j + 1][i + 1] +
                Ac[k + 1][j - 1][i - 1] + Ac[k + 1][j - 1][i + 1] +
                Ac[k + 1][j + 1][i - 1] + Ac[k + 1][j + 1][i + 1]) +
        0.1 * (Ac[k - 1][j - 1][i] + Ac[k - 1][j][i - 1] +
               Ac[k - 1][j][i + 1] + Ac[k - 1][j + 1][i] +
               Ac[k][j - 1][i - 1] + Ac[k][j - 1][i + 1] +
               Ac[k][j + 1][i - 1] + Ac[k][j + 1][i + 1] +
               Ac[k + 1][j - 1][i] + Ac[k + 1][j][i - 1] +
               Ac[k + 1][j][i + 1] + Ac[k + 1][j + 1][i]) +
        0.46 * (Ac[k - 1][j][i] + Ac[k][j - 1][i] + Ac[k][j][i - 1] +
                Ac[k + 1][j][i] + Ac[k][j + 1][i] + Ac[k][j][i + 1]) +
        4.26 * Ac[k][j][i]
               )));
}
__global__ void chebyshev_naive_bricks1(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  unsigned i = threadIdx.x;
  unsigned j = threadIdx.y;
  unsigned k = threadIdx.z;
  out[b][k][j][i] = 
    Ac[b][k][j][i] + c[0] * (Ac[b][k][j][i] + Ap[b][k][j][i]) +
    c[1] * Dinv[b][k][j][i] *
    (RHS[b][k][j][i] +
    (Ac[b][k][j][i] + 
    c[2] * 
      (
        0.03 * (Ac[b][k - 1][j - 1][i - 1] + Ac[b][k - 1][j - 1][i + 1] +
                Ac[b][k - 1][j + 1][i - 1] + Ac[b][k - 1][j + 1][i + 1] +
                Ac[b][k + 1][j - 1][i - 1] + Ac[b][k + 1][j - 1][i + 1] +
                Ac[b][k + 1][j + 1][i - 1] + Ac[b][k + 1][j + 1][i + 1]) +
        0.1 * (Ac[b][k - 1][j - 1][i] + Ac[b][k - 1][j][i - 1] +
               Ac[b][k - 1][j][i + 1] + Ac[b][k - 1][j + 1][i] +
               Ac[b][k][j - 1][i - 1] + Ac[b][k][j - 1][i + 1] +
               Ac[b][k][j + 1][i - 1] + Ac[b][k][j + 1][i + 1] +
               Ac[b][k + 1][j - 1][i] + Ac[b][k + 1][j][i - 1] +
               Ac[b][k + 1][j][i + 1] + Ac[b][k + 1][j + 1][i]) +
        0.46 * (Ac[b][k - 1][j][i] + Ac[b][k][j - 1][i] + Ac[b][k][j][i - 1] +
                Ac[b][k + 1][j][i] + Ac[b][k][j + 1][i] + Ac[b][k][j][i + 1]) +
        4.26 * Ac[b][k][j][i]
               )));
}
}
__global__ void chebyshev_codegen_bricks1(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/chebyshev/intermediate_gen/chebyshev1.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
#define Ac(a, b, c) Ac_arr[c][b][a]
#define Ap(a, b, c) Ap_arr[c][b][a]
#define Dinv(a, b, c) Dinv_arr[c][b][a]
#define RHS(a, b, c) RHS_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void chebyshev_codegen1(bElem (*Ac_arr)[STRIDE1][STRIDE0], bElem (*Ap_arr)[STRIDE1][STRIDE0], bElem (*Dinv_arr)[STRIDE1][STRIDE0], bElem (*RHS_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
  long k = OFF2 + (blockIdx.z * TILE2);
  long j = OFF1 + (blockIdx.y * TILE1);
  long i = OFF0 + (blockIdx.x * TILE0);
  tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/chebyshev/intermediate_gen/chebyshev1.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef Ac
#undef Ap
#undef Dinv
#undef RHS
#undef out
