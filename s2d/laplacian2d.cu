#include "hip/hip_runtime.h"

#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// #include "../out/laplacian-stencils.h"

// $START naive
__global__ void laplacian2d_naive(bElem (*in)[STRIDE0], bElem (*out)[STRIDE0], bElem* dev_coeff) {
    const size_t radius = $SIZE;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;

    bElem temp = dev_coeff[0] * in[j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[j][i + a] + in[j + a][i] +
            in[j][i - a] + in[j - a][i]);
    }
    out[j][i] = temp;
}
// $END naive

// $START naive-bricks
__global__ void laplacian2d_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    bOut[b][j][i] = dev_coeff[0] * bIn[b][j][i];

    const size_t radius = $SIZE;
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][j][i] += dev_coeff[a] * (
            bIn[b][j][i + a] + bIn[b][j + a][i] +
            bIn[b][j][i - a] + bIn[b][j - a][i]
        );
    }
}
// $END naive-bricks

// $START codegen-bricks
__global__ void laplacian2d_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem *dev_coeff) {
    unsigned b = grid[blockIdx.y + GB1][blockIdx.x + GB0];
    brick("$PYTHON", VSVEC, (TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks


// $START codegen
#define bIn(a, b) arr_in[c][b]
#define bOut(a, b) arr_out[c][b]

__global__ void laplacian_codegen(bElem (*arr_in)[STRIDE0], bElem (*arr_out)[STRIDE0], bElem *dev_coeff) {
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("$PYTHON", VSVEC, (TILE1, VECSIZE), ("j", "i"), (1, VECSIZE));
}

#undef bIn
#undef bOut
// $END codegen
