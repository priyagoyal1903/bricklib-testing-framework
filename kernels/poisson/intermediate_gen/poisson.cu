#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"
#include "../../../gen/consts.h"
#include "./poisson.h"
#include <brick-hip.h>

__global__ void poisson_naive2(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen2(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson2.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef in
#undef out
__global__ void poisson_naive_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks2(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson2.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void poisson_naive3(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen3(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson3.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef in
#undef out
__global__ void poisson_naive_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks3(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson3.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
__global__ void poisson_naive5(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;
    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen5(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * TILE0);
    tile("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson5.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef in
#undef out
__global__ void poisson_naive_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
__global__ void poisson_codegen_bricks5(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
  brick("/autofs/nccs-svm1_home1/priyagoyal/bricklib-testing-framework/kernels/poisson/intermediate_gen/poisson5.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
